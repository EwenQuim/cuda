#include "hip/hip_runtime.h"
/*********************************************************************************/
/* Matrix product program for a multi-core CPU and for a many-core GPU           */
/* S. Vialle - November 2021                                                     */
/*********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "main.h"
#include "gpu.h"


/*-------------------------------------------------------------------------------*/
/* GPU symbols and global vars                                                   */
/*-------------------------------------------------------------------------------*/
// Symbols used by all kernels
__device__ T_real GPU_A[SIZE][SIZE];
__device__ T_real GPU_B[SIZE][SIZE];
__device__ T_real GPU_C[SIZE][SIZE];

// New Symbol and vars to call Cublas lib.
__device__ T_real GPU_Ctmp[SIZE][SIZE];   // New matrix buffer

T_real *AdrGPU_A = NULL;                  // Adresses of the symbols
T_real *AdrGPU_B = NULL;
T_real *AdrGPU_C = NULL;
T_real *AdrGPU_Ctmp = NULL; 

hipblasHandle_t cublasHandle;              // Handle on the Cublas lib.


/*-------------------------------------------------------------------------------*/
/* Init and finalize the GPU device.                                             */
/*-------------------------------------------------------------------------------*/
void gpuInit(void)
{
  hipInit(0);
  
  // Extract address of GPU matrix "symbols"
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_A,GPU_A),"GPU_A adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_B,GPU_B),"GPU_B adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_C,GPU_C),"GPU_C adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_Ctmp,GPU_Ctmp),"GPU_Ctmp adr extraction");
  
  // Turn CPU arrays A, B and C into "pinned" memory areas
  CHECK_CUDA_SUCCESS(hipHostRegister(A,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(B,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(C,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the C CPU array");
  
  // Initialize CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasCreate(&cublasHandle), "Init of the CUBLAS lib handle"); 
}


void gpuFinalize(void)
{
  // Turn "pinned" CPU arrays into std array
  CHECK_CUDA_SUCCESS(hipHostUnregister(A),
                     "Turning into std memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(B),
                     "Turning into std memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(C),
                     "Turning into std memory the C CPU array");

  // Free CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasDestroy(cublasHandle), "Free the CUBLAS lib");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                   */
/*-------------------------------------------------------------------------------*/
void gpuSetDataOnGPU(void)
{
  // Set GPU_A symbol
  //CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(...),
  //                   "Transfer A-->GPU_A");

  // Set GPU_B symbol
  // ...
}


/*-------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                        */
/*-------------------------------------------------------------------------------*/
void gpuGetResultOnCPU(void)
{
  // Get GPU_C symbol
  // ...
}


/*-------------------------------------------------------------------------------*/
/* Transposition kernel using global memory and registers.                       */
/*-------------------------------------------------------------------------------*/
__global__ void TransposeKernel_v0(T_real *MT, T_real *M, int mLig, int nCol)
{
 int lig = threadIdx.y + blockIdx.y*BLOCK_SIZE_XY_KT0;
 int col = threadIdx.x + blockIdx.x*BLOCK_SIZE_XY_KT0;
 
 if (lig < mLig && col < nCol)
   MT[col*mLig + lig] = M[lig*nCol + col];
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 1D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v0(void)
{
  // Index computations
  //int lig = ...
  //int col = ...
  //T_real res = 0.0;

  // Matrix product computation
  // ...
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 2D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v1(void)
{
 // Index computations
 //int lig = ...
 //int col = ..

 // Matrix product computation
 //...
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU.                                        */
/*-------------------------------------------------------------------------------*/
void gpuProduct(gkid_t kid)
{
 dim3 Dg = {0,0,0};   // Grid descriptor
 dim3 Db = {0,0,0};   // Block descriptor
 
 //T_real alpha;        // When using CUBLAS
 //T_real beta;         // When using CUBLAS

 switch(kid) {

 case GK0 : // Kernel v0 - 1D kernel using only resgisters and cache with generic matrix size
   // - init the grid of blocs
   //Db.x = ;
   //Db.y = ;
   //Db.z = ;
   //Dg.x = ;
   //Dg.y = ;
   //Dg.z = ;
   // - run the Grid of Blocs of threads
   //MatrixProductKernel_v0<<<Dg,Db>>>();
   break;

 case GK1 : // kernel v1 : 2D kernel using only registers and cache with generic matrix size
   break;

 case GK2 : // kernel v2 : 2D kernel using the shared memories
   break;
  
 case GK3 : // kernel v3 : 2D kernel using the shared memories with generic matrix size
   break;

 case GK4 : // calling cublas gemm & user-defined transpose kernel
   break;
   
 case GK5 : // Calling cublas gemm & cublas geam kernels
   break;

 case GK6 : // Calling cublas gemm, using matrix math properties
   break;

 case GK7 : // Calling cublas gemmEx, using Tensor cores
   break;

 case GK8 : // Free
   break;

 default :
   fprintf(stderr,"Unknown GPU kernel!");
   exit(EXIT_FAILURE);
 } // End of switch
}




