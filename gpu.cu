#include "hip/hip_runtime.h"
/*********************************************************************************/
/* Matrix product program for a multi-core CPU and for a many-core GPU           */
/* S. Vialle - November 2021                                                     */
/*********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "main.h"
#include "gpu.h"

/*-------------------------------------------------------------------------------*/
/* GPU symbols and global vars                                                   */
/*-------------------------------------------------------------------------------*/
// Symbols used by all kernels
__device__ T_real GPU_A[SIZE][SIZE];
__device__ T_real GPU_B[SIZE][SIZE];
__device__ T_real GPU_C[SIZE][SIZE];

// New Symbol and vars to call Cublas lib.
__device__ T_real GPU_Ctmp[SIZE][SIZE]; // New matrix buffer

T_real *AdrGPU_A = NULL; // Adresses of the symbols
T_real *AdrGPU_B = NULL;
T_real *AdrGPU_C = NULL;
T_real *AdrGPU_Ctmp = NULL;

hipblasHandle_t cublasHandle; // Handle on the Cublas lib.

/*-------------------------------------------------------------------------------*/
/* Init and finalize the GPU device.                                             */
/*-------------------------------------------------------------------------------*/
void gpuInit(void)
{
  hipInit(0);

  // Extract address of GPU matrix "symbols"
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_A, GPU_A), "GPU_A adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_B, GPU_B), "GPU_B adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_C, GPU_C), "GPU_C adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_Ctmp, GPU_Ctmp), "GPU_Ctmp adr extraction");

  // Turn CPU arrays A, B and C into "pinned" memory areas
  CHECK_CUDA_SUCCESS(hipHostRegister(A, SIZE * SIZE * sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(B, SIZE * SIZE * sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(C, SIZE * SIZE * sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the C CPU array");

  // Initialize CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasCreate(&cublasHandle), "Init of the CUBLAS lib handle");
}

void gpuFinalize(void)
{
  // Turn "pinned" CPU arrays into std array
  CHECK_CUDA_SUCCESS(hipHostUnregister(A),
                     "Turning into std memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(B),
                     "Turning into std memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(C),
                     "Turning into std memory the C CPU array");

  // Free CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasDestroy(cublasHandle), "Free the CUBLAS lib");
}

/*-------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                   */
/*-------------------------------------------------------------------------------*/
void gpuSetDataOnGPU(void)
{
  // Set GPU_A symbol
  CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(GPU_A), &A[0], SIZE * SIZE * sizeof(T_real), 0, hipMemcpyHostToDevice),
                     "Transfer A-->GPU_A");

  // Set GPU_B symbol
  CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(GPU_B), &B[0], SIZE * SIZE * sizeof(T_real), 0, hipMemcpyHostToDevice),
                     "Transfer B-->GPU_B");
}

/*-------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                        */
/*-------------------------------------------------------------------------------*/
void gpuGetResultOnCPU(void)
{
  // Get GPU_C symbol
  CHECK_CUDA_SUCCESS(hipMemcpyFromSymbol(&C[0], HIP_SYMBOL(GPU_C),
                                          SIZE * SIZE * sizeof(T_real), 0,
                                          hipMemcpyDeviceToHost),
                     "Transfer GPU_C-->C");
}

/*-------------------------------------------------------------------------------*/
/* Transposition kernel using global memory and registers.                       */
/*-------------------------------------------------------------------------------*/
__global__ void TransposeKernel_v0(T_real *MT, T_real *M, int mLig, int nCol)
{
  int lig = threadIdx.y + blockIdx.y * BLOCK_SIZE_XY_KT0;
  int col = threadIdx.x + blockIdx.x * BLOCK_SIZE_XY_KT0;

  if (lig < mLig && col < nCol)
    MT[col * mLig + lig] = M[lig * nCol + col];
}

/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 1D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v0(void)
{
  // Index computations
  int lig = blockIdx.y;
  int col = blockIdx.x * BLOCK_SIZE_X_K0 + threadIdx.x;
  if (lig >= SIZE || col >= SIZE)
  {
    return;
  }
  T_real res = 0.0;

  // Matrix product computation
  for (int i = 0; i < SIZE; i++)
  {
    res += GPU_A[lig][i] * GPU_B[i][col];
  }
  GPU_C[lig][col] = res;
}

/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 2D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v1(void)
{
  // Index computations
  //int lig = ...
  //int col = ..

  // Matrix product computation
  //...
}

/*-------------------------------------------------------------------------------*/
/* 2D kernel using the shared memories              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v2(void)
{
  __shared__ T_real shdataA[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  __shared__ T_real shdataB[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  __shared__ T_real shdataC[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];

  // Index computations
  int lig = blockIdx.y * BLOCK_SIZE_XY_K2 + threadIdx.y;
  int col = blockIdx.x * BLOCK_SIZE_XY_K2 + threadIdx.x;
  if (lig >= SIZE || col >= SIZE)
  {
    return;
  }
  shdataC[threadIdx.y][threadIdx.x] = 0.0;

  for (int blockNum = 0; blockNum < SIZE / BLOCK_SIZE_XY_K2; blockNum++)
  {
    shdataA[threadIdx.y][threadIdx.x] = GPU_A[lig][blockNum * BLOCK_SIZE_XY_K2 + threadIdx.x];
    shdataB[threadIdx.y][threadIdx.x] = GPU_B[blockNum * BLOCK_SIZE_XY_K2 + threadIdx.y][col];
    __syncthreads();

    // Matrix product computation
    for (int i = 0; i < BLOCK_SIZE_XY_K2; i++)
    {
      shdataC[threadIdx.y][threadIdx.x] += shdataA[threadIdx.y][i] * shdataB[i][threadIdx.x];
    }
  }
  GPU_C[lig][col] = shdataC[threadIdx.y][threadIdx.x];
}

/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU.                                        */
/*-------------------------------------------------------------------------------*/
void gpuProduct(gkid_t kid)
{
  dim3 Dg = {0, 0, 0}; // Grid descriptor
  dim3 Db = {0, 0, 0}; // Block descriptor

  //T_real alpha;        // When using CUBLAS
  //T_real beta;         // When using CUBLAS

  switch (kid)
  {
  case GK0: // Kernel v0 - 1D kernel using only resgisters and cache with generic matrix size
    // - init the grid of blocs
    Db.x = BLOCK_SIZE_X_K0;
    Db.y = 1;
    Db.z = 1;
    Dg.x = (SIZE - 1) / BLOCK_SIZE_X_K0 + 1;
    Dg.y = SIZE;
    Dg.z = 1;
    // - run the Grid of Blocs of threads
    MatrixProductKernel_v0<<<Dg, Db>>>();
    break;

  case GK1: // kernel v1 : 2D kernel using only registers and cache with generic matrix size
    break;

  case GK2: // kernel v2 : 2D kernel using the shared memories
    // - init the grid of blocs
    Db.x = BLOCK_SIZE_XY_K2;
    Db.y = BLOCK_SIZE_XY_K2;
    Db.z = 1;
    Dg.x = (SIZE - 1) / BLOCK_SIZE_XY_K2 + 1;
    Dg.y = (SIZE - 1) / BLOCK_SIZE_XY_K2 + 1;
    Dg.z = 1;
    // - run the Grid of Blocs of threads
    MatrixProductKernel_v2<<<Dg, Db>>>();
    break;

  case GK3: // kernel v3 : 2D kernel using the shared memories with generic matrix size
    break;

  case GK4: // calling cublas gemm & user-defined transpose kernel
    break;

  case GK5: // Calling cublas gemm & cublas geam kernels
    break;

  case GK6: // Calling cublas gemm, using matrix math properties
    break;

  case GK7: // Calling cublas gemmEx, using Tensor cores
    break;

  case GK8: // Free
    break;

  default:
    fprintf(stderr, "Unknown GPU kernel!");
    exit(EXIT_FAILURE);
  } // End of switch
}
